#include "hip/hip_runtime.h"
#include "f5c.h"
#include <assert.h>
#include "f5cmisc.cuh"

#ifdef ALIGN_KERNEL_SLICED

//#define DEBUG_ESTIMATED_SCALING 1
//#define DEBUG_RECALIB_SCALING 1
//#define DEBUG_ADAPTIVE 1


//todo : can make more efficient using bit encoding
//todo : is inlining correct?
__forceinline__ __device__  uint32_t get_rank(char base) {
    if (base == 'A') { //todo: do we neeed simple alpha?
        return 0;
    } else if (base == 'C') {
        return 1;
    } else if (base == 'G') {
        return 2;
    } else if (base == 'T') {
        return 3;
    } else {
        //WARNING("A None ACGT base found : %c", base); //todo : fix this in gpu code
        return 0;
    }
}

// return the lexicographic rank of the kmer amongst all strings of
// length k for this alphabet
__forceinline__ __device__  uint32_t get_kmer_rank(const char* str, uint32_t k) {
    //uint32_t p = 1;
    uint32_t r = 0;

    // from last base to first
    for (uint32_t i = 0; i < k; ++i) {
        //r += rank(str[k - i - 1]) * p;
        //p *= size();
        r += get_rank(str[k - i - 1]) << (i << 1);
    }
    return r;
}

//copy a kmer from a reference
__forceinline__ __device__ void kmer_cpy(char* dest, char* src, uint32_t k) {
    uint32_t i = 0;
    for (i = 0; i < k; i++) {
        dest[i] = src[i];
    }
    dest[i] = '\0';
}

#define log_inv_sqrt_2pi  -0.918938f // Natural logarithm

__forceinline__ __device__ float
log_normal_pdf(float x, float gp_mean, float gp_stdv, float gp_log_stdv) {
    /*INCOMPLETE*/
    //float log_inv_sqrt_2pi = -0.918938f; // Natural logarithm
    float a = (x - gp_mean) / gp_stdv;
    return log_inv_sqrt_2pi - gp_log_stdv + (-0.5f * a * a);
    // return 1;
}

__forceinline__ __device__ float
log_probability_match_r9(scalings_t scaling, model_t* models, event_t* event,
                         int event_idx, uint32_t kmer_rank) {
    // event level mean, scaled with the drift value
    //strand = 0;
 #ifdef DEBUG_ADAPTIVE   
    assert(kmer_rank < 4096);
 #endif   
    //float level = read.get_drift_scaled_level(event_idx, strand);

    //float time =
    //    (events.event[event_idx].start - events.event[0].start) / sample_rate;
    float unscaledLevel = event[event_idx].mean;
    float scaledLevel = unscaledLevel;
    //float scaledLevel = unscaledLevel - time * scaling.shift;

    //fprintf(stderr, "level %f\n",scaledLevel);
    //GaussianParameters gp = read.get_scaled_gaussian_from_pore_model_state(pore_model, strand, kmer_rank);
    float gp_mean =
        scaling.scale * models[kmer_rank].level_mean + scaling.shift;
    float gp_stdv = models[kmer_rank].level_stdv * 1; //scaling.var = 1;
    // float gp_stdv = 0;
    // float gp_log_stdv = models[kmer_rank].level_log_stdv + scaling.log_var;
    // if(models[kmer_rank].level_stdv <0.01 ){
    // 	fprintf(stderr,"very small std dev %f\n",models[kmer_rank].level_stdv);
    // }
    float gp_log_stdv =
        log(models[kmer_rank].level_stdv + 0); // scaling.log_var = log(1)=0;

    float lp = log_normal_pdf(scaledLevel, gp_mean, gp_stdv, gp_log_stdv);
    return lp;
}

#define event_kmer_to_band(ei, ki) (ei + 1) + (ki + 1)
#define band_event_to_offset(bi, ei) band_lower_left[bi].event_idx - (ei)
#define band_kmer_to_offset(bi, ki) (ki) - band_lower_left[bi].kmer_idx
#define is_offset_valid(offset) (offset) >= 0 && (offset) < bandwidth
#define event_at_offset(bi, offset) band_lower_left[(bi)].event_idx - (offset)
#define kmer_at_offset(bi, offset) band_lower_left[(bi)].kmer_idx + (offset)

#define move_down(curr_band)                                                   \
    { curr_band.event_idx + 1, curr_band.kmer_idx }
#define move_right(curr_band)                                                  \
    { curr_band.event_idx, curr_band.kmer_idx + 1 }

#define MIN(a, b) (((a) < (b)) ? (a) : (b))
#define MAX(a, b) (((a) > (b)) ? (a) : (b))

#define BAND_ARRAY(r, c) ( bands[((r)*(ALN_BANDWIDTH)+(c))] )
#define TRACE_ARRAY(r, c) ( trace[((r)*(ALN_BANDWIDTH)+(c))] )

#define FROM_D  0
#define FROM_U  1
#define FROM_L  2

#define min_average_log_emission  -5.0
#define max_gap_threshold  50
#define bandwidth  ALN_BANDWIDTH
#define half_bandwidth  ALN_BANDWIDTH/2

#define epsilon 1e-10

__global__ void adaptive_align_inner(float *bands,uint8_t *trace,EventKmerPair* band_lower_left,
    int min_offset, int max_offset, int band_idx, int32_t* kmer_ranks,
    scalings_t scaling, model_t* models, event_t* events,double p_stay){
    
    //double epsilon = 1e-10;
    double lp_skip = log(epsilon);
    double lp_stay = log(p_stay);
    double lp_step = log(1.0 - exp(lp_skip) - exp(lp_stay));
    //double lp_trim = log(0.01);

    int tid=blockIdx.x*blockDim.x+threadIdx.x;
    //for (int offset = min_offset; offset < max_offset; ++offset) {
    if(tid < (max_offset-min_offset)) {
        int offset=min_offset+tid;

        int event_idx = event_at_offset(band_idx, offset);
        int kmer_idx = kmer_at_offset(band_idx, offset);

        int32_t kmer_rank = kmer_ranks[kmer_idx];

        int offset_up = band_event_to_offset(band_idx - 1, event_idx - 1);
        int offset_left = band_kmer_to_offset(band_idx - 1, kmer_idx - 1);
        int offset_diag = band_kmer_to_offset(band_idx - 2, kmer_idx - 1);

#ifdef DEBUG_ADAPTIVE
        // verify loop conditions
        assert(kmer_idx >= 0 && kmer_idx < n_kmers);
        assert(event_idx >= 0 && event_idx < n_events);
        assert(offset_diag ==
               band_event_to_offset(band_idx - 2, event_idx - 1));
        assert(offset_up - offset_left == 1);
        assert(offset >= 0 && offset < bandwidth);
#endif //DEBUG_ADAPTIVE

        float up = is_offset_valid(offset_up)
                       ? BAND_ARRAY(band_idx - 1,offset_up)
                       : -INFINITY;
        float left = is_offset_valid(offset_left)
                         ? BAND_ARRAY(band_idx - 1,offset_left)
                         : -INFINITY;
        float diag = is_offset_valid(offset_diag)
                         ? BAND_ARRAY(band_idx - 2,offset_diag)
                         : -INFINITY;

        float lp_emission = log_probability_match_r9(
            scaling, models, events, event_idx, kmer_rank);
        //fprintf(stderr, "lp emiision : %f , event idx %d, kmer rank %d\n", lp_emission,event_idx,kmer_rank);
        float score_d = diag + lp_step + lp_emission;
        float score_u = up + lp_stay + lp_emission;
        float score_l = left + lp_skip;

        float max_score = score_d;
        uint8_t from = FROM_D;

        max_score = score_u > max_score ? score_u : max_score;
        from = max_score == score_u ? FROM_U : from;
        max_score = score_l > max_score ? score_l : max_score;
        from = max_score == score_l ? FROM_L : from;

#ifdef DEBUG_ADAPTIVE
        fprintf(stderr,
                "[adafill] offset-up: %d offset-diag: %d offset-left: %d\n",
                offset_up, offset_diag, offset_left);
        fprintf(stderr, "[adafill] up: %.2lf diag: %.2lf left: %.2lf\n", up,
                diag, left);
        fprintf(stderr,
                "[adafill] bi: %d o: %d e: %d k: %d s: %.2lf f: %d emit: "
                "%.2lf\n",
                band_idx, offset, event_idx, kmer_idx, max_score, from,
                lp_emission);
#endif //DEBUG_ADAPTIVE
        BAND_ARRAY(band_idx,offset) = max_score;
        TRACE_ARRAY(band_idx,offset) = from;
        //fills += 1;
    }

}

__forceinline__ __device__ void align_single_pre(AlignedPair* out_2, char* sequence,
                                int32_t sequence_len, event_t* events,
                                int32_t n_event, model_t* models,
                                scalings_t scaling,int32_t* kmer_ranks,float *bands,uint8_t *trace, EventKmerPair* band_lower_left) {
    //fprintf(stderr, "%s\n", sequence);
    //fprintf(stderr, "Scaling %f %f", scaling.scale, scaling.shift);

    //size_t strand_idx = 0;
    //size_t k = 6;

    // size_t n_events = events[strand_idx].n;
    int32_t n_events = n_event;
    int32_t n_kmers = sequence_len - KMER_SIZE + 1;
    //fprintf(stderr,"n_kmers : %d\n",n_kmers);
    // backtrack markers
    //const uint8_t FROM_D = 0;
    //const uint8_t FROM_U = 1;
    //const uint8_t FROM_L = 2;

    // qc
    //double min_average_log_emission = -5.0;
    //int max_gap_threshold = 50;

    // banding
    //int bandwidth = ALN_BANDWIDTH;
    //half_bandwidth = bandwidth / 2;

    // transition penalties
    float events_per_kmer = (float)n_events / n_kmers;
    float p_stay = 1 - (1 / (events_per_kmer + 1));

    // setting a tiny skip penalty helps keep the true alignment within the adaptive band
    // this was empirically determined
    //double epsilon = 1e-10;
    double lp_skip = log(epsilon);
    double lp_stay = log(p_stay);
    double lp_step = log(1.0 - exp(lp_skip) - exp(lp_stay));
    double lp_trim = log(0.01);

    // dp matrix
    int32_t n_rows = n_events + 1;
    int32_t n_cols = n_kmers + 1;
    int32_t n_bands = n_rows + n_cols;

    // Initialize

    // Precompute k-mer ranks to avoid doing this in the inner loop
    //size_t* kmer_ranks = (size_t*)malloc(sizeof(size_t) * n_kmers);
    //MALLOC_CHK(kmer_ranks); //todo : fix these to error check

    for (int32_t i = 0; i < n_kmers; ++i) {
        //>>>>>>>>> New replacement begin
        char* substring = &sequence[i];
        kmer_ranks[i] = get_kmer_rank(substring, KMER_SIZE);
        //<<<<<<<<< New replacement over
    }

    //float** bands = (float**)malloc(sizeof(float*) * n_bands);
    //MALLOC_CHK(bands);
    //uint8_t** trace = (uint8_t**)malloc(sizeof(uint8_t*) * n_bands);
    //MALLOC_CHK(trace);

    for (int32_t i = 0; i < n_bands; i++) {
        //bands[i] = (float*)malloc(sizeof(float) * bandwidth);
        //MALLOC_CHK(bands[i]);
        //trace[i] = (uint8_t*)malloc(sizeof(uint8_t) * bandwidth);
        //MALLOC_CHK(trace[i]);

        for (int j = 0; j < bandwidth; j++) {
            BAND_ARRAY(i,j) = -INFINITY;
            TRACE_ARRAY(i,j) = 0;
        }
    }

    // Keep track of the event/kmer index for the lower left corner of the band
    // these indices are updated at every iteration to perform the adaptive banding
    // Only the first two bands have their coordinates initialized, the rest are computed adaptively

    // struct EventKmerPair {
    //     int event_idx;
    //     int kmer_idx;
    // };
    //>>>>>>>>>>>>>>>>>New Replacement Begin
    //struct EventKmerPair* band_lower_left =
    //    (struct EventKmerPair*)malloc(sizeof(struct EventKmerPair) * n_bands);
    //MALLOC_CHK(band_lower_left);
    //std::vector<EventKmerPair> band_lower_left(n_bands);
    //<<<<<<<<<<<<<<<<<New Replacement over

    // initialize range of first two bands
    band_lower_left[0].event_idx = half_bandwidth - 1;
    band_lower_left[0].kmer_idx = -1 - half_bandwidth;
    band_lower_left[1] = move_down(band_lower_left[0]);

    int start_cell_offset = band_kmer_to_offset(0, -1);
    assert(is_offset_valid(start_cell_offset));
    assert(band_event_to_offset(0, -1) == start_cell_offset);
    BAND_ARRAY(0,start_cell_offset) = 0.0f;

    // band 1: first event is trimmed
    int first_trim_offset = band_event_to_offset(1, 0);
    assert(kmer_at_offset(1, first_trim_offset) == -1);
    assert(is_offset_valid(first_trim_offset));
    BAND_ARRAY(1,first_trim_offset) = lp_trim;
    TRACE_ARRAY(1,first_trim_offset) = FROM_U;

    //int fills = 0;
#ifdef DEBUG_ADAPTIVE
    fprintf(stderr, "[trim] bi: %d o: %d e: %d k: %d s: %.2lf\n", 1,
            first_trim_offset, 0, -1, BAND_ARRAY(1,first_trim_offset);
#endif

}

__forceinline__ __device__ void align_single(AlignedPair* out_2, char* sequence,
                                int32_t sequence_len, event_t* events,
                                int32_t n_event, model_t* models,
                                scalings_t scaling,int32_t* kmer_ranks,float *bands,uint8_t *trace, EventKmerPair* band_lower_left) {

    //fprintf(stderr, "%s\n", sequence);
    //fprintf(stderr, "Scaling %f %f", scaling.scale, scaling.shift);

    //size_t strand_idx = 0;
    //size_t k = 6;

    // size_t n_events = events[strand_idx].n;
    int32_t n_events = n_event;
    int32_t n_kmers = sequence_len - KMER_SIZE + 1;
    //fprintf(stderr,"n_kmers : %d\n",n_kmers);
    // backtrack markers
    //const uint8_t FROM_D = 0;
    //const uint8_t FROM_U = 1;
    //const uint8_t FROM_L = 2;

    // qc
    //double min_average_log_emission = -5.0;
    //int max_gap_threshold = 50;

    // banding
    //int bandwidth = ALN_BANDWIDTH;
    //half_bandwidth = bandwidth / 2;

    // transition penalties
    float events_per_kmer = (float)n_events / n_kmers;
    float p_stay = 1 - (1 / (events_per_kmer + 1));

    // setting a tiny skip penalty helps keep the true alignment within the adaptive band
    // this was empirically determined
    //double epsilon = 1e-10;
    double lp_skip = log(epsilon);
    double lp_stay = log(p_stay);
    double lp_step = log(1.0 - exp(lp_skip) - exp(lp_stay));
    double lp_trim = log(0.01);

    // dp matrix
    int32_t n_rows = n_events + 1;
    int32_t n_cols = n_kmers + 1;
    int32_t n_bands = n_rows + n_cols;                                    

    // fill in remaining bands
    for (int32_t band_idx = 2; band_idx < n_bands; ++band_idx) {
        // Determine placement of this band according to Suzuki's adaptive algorithm
        // When both ll and ur are out-of-band (ob) we alternate movements
        // otherwise we decide based on scores
        float ll = BAND_ARRAY((band_idx - 1), 0);
        float ur = BAND_ARRAY((band_idx - 1),(bandwidth - 1));
        bool ll_ob = ll == -INFINITY;
        bool ur_ob = ur == -INFINITY;

        bool right = false;
        if (ll_ob && ur_ob) {
            right = band_idx % 2 == 1;
        } else {
            right = ll < ur; // Suzuki's rule
        }

        if (right) {
            band_lower_left[band_idx] =
                move_right(band_lower_left[band_idx - 1]);
        } else {
            band_lower_left[band_idx] =
                move_down(band_lower_left[band_idx - 1]);
        }
        // If the trim state is within the band, fill it in here
        int trim_offset = band_kmer_to_offset(band_idx, -1);
        if (is_offset_valid(trim_offset)) {
            int32_t event_idx = event_at_offset(band_idx, trim_offset);
            if (event_idx >= 0 && event_idx < n_events) {
                BAND_ARRAY(band_idx,trim_offset) = lp_trim * (event_idx + 1);
                TRACE_ARRAY(band_idx,trim_offset) = FROM_U;
            } else {
                BAND_ARRAY(band_idx,trim_offset) = -INFINITY;
            }
        }

        // Get the offsets for the first and last event and kmer
        // We restrict the inner loop to only these values
        int kmer_min_offset = band_kmer_to_offset(band_idx, 0);
        int kmer_max_offset = band_kmer_to_offset(band_idx, n_kmers);
        int event_min_offset = band_event_to_offset(band_idx, n_events - 1);
        int event_max_offset = band_event_to_offset(band_idx, -1);

        int min_offset = MAX(kmer_min_offset, event_min_offset);
        min_offset = MAX(min_offset, 0);

        int max_offset = MIN(kmer_max_offset, event_max_offset);
        max_offset = MIN(max_offset, bandwidth);

#ifdef DYNAMIC_PARALLELISM
        if(max_offset-min_offset > DYNAMIC_THRESH){
            int32_t BLOCK_LEN = DYNAMIC_BLOCK_LEN;
            int grid=((max_offset-min_offset + BLOCK_LEN - 1) / BLOCK_LEN);
            int block=(BLOCK_LEN);
            adaptive_align_inner<<<grid, block>>>(bands,trace,band_lower_left,min_offset, max_offset, band_idx, kmer_ranks,scaling, models, events,p_stay);        
            hipDeviceSynchronize();
            hipError_t code = hipGetLastError();
            if (code != hipSuccess) {
                printf("grid %d, block %d\n",grid,block);
                printf("Cuda error: %s \n in file : %s line number : %lu\n",
                        hipGetErrorString(code), __FILE__, __LINE__);
                return ; //todo : generalise
            }
        }
        else{
#endif  //dynamic paralellism 

        for (int offset = min_offset; offset < max_offset; ++offset) {
            int event_idx = event_at_offset(band_idx, offset);
            int kmer_idx = kmer_at_offset(band_idx, offset);

            int32_t kmer_rank = kmer_ranks[kmer_idx];

            int offset_up = band_event_to_offset(band_idx - 1, event_idx - 1);
            int offset_left = band_kmer_to_offset(band_idx - 1, kmer_idx - 1);
            int offset_diag = band_kmer_to_offset(band_idx - 2, kmer_idx - 1);

#ifdef DEBUG_ADAPTIVE
            // verify loop conditions
            assert(kmer_idx >= 0 && kmer_idx < n_kmers);
            assert(event_idx >= 0 && event_idx < n_events);
            assert(offset_diag ==
                   band_event_to_offset(band_idx - 2, event_idx - 1));
            assert(offset_up - offset_left == 1);
            assert(offset >= 0 && offset < bandwidth);
#endif //DEBUG_ADAPTIVE

            float up = is_offset_valid(offset_up)
                           ? BAND_ARRAY(band_idx - 1,offset_up)
                           : -INFINITY;
            float left = is_offset_valid(offset_left)
                             ? BAND_ARRAY(band_idx - 1,offset_left)
                             : -INFINITY;
            float diag = is_offset_valid(offset_diag)
                             ? BAND_ARRAY(band_idx - 2,offset_diag)
                             : -INFINITY;

            float lp_emission = log_probability_match_r9(
                scaling, models, events, event_idx, kmer_rank);
            //fprintf(stderr, "lp emiision : %f , event idx %d, kmer rank %d\n", lp_emission,event_idx,kmer_rank);
            float score_d = diag + lp_step + lp_emission;
            float score_u = up + lp_stay + lp_emission;
            float score_l = left + lp_skip;

            float max_score = score_d;
            uint8_t from = FROM_D;

            max_score = score_u > max_score ? score_u : max_score;
            from = max_score == score_u ? FROM_U : from;
            max_score = score_l > max_score ? score_l : max_score;
            from = max_score == score_l ? FROM_L : from;

#ifdef DEBUG_ADAPTIVE
            fprintf(stderr,
                    "[adafill] offset-up: %d offset-diag: %d offset-left: %d\n",
                    offset_up, offset_diag, offset_left);
            fprintf(stderr, "[adafill] up: %.2lf diag: %.2lf left: %.2lf\n", up,
                    diag, left);
            fprintf(stderr,
                    "[adafill] bi: %d o: %d e: %d k: %d s: %.2lf f: %d emit: "
                    "%.2lf\n",
                    band_idx, offset, event_idx, kmer_idx, max_score, from,
                    lp_emission);
#endif //DEBUG_ADAPTIVE
            BAND_ARRAY(band_idx,offset) = max_score;
            TRACE_ARRAY(band_idx,offset) = from;
            //fills += 1;
        }

#ifdef DYNAMIC_PARALLELISM
        }
#endif  //dynamic paralellism     

    }

}

__forceinline__ __device__ int32_t align_single_post(AlignedPair* out_2, char* sequence,
    int32_t sequence_len, event_t* events,
    int32_t n_event, model_t* models,
    scalings_t scaling,int32_t* kmer_ranks,float *bands,uint8_t *trace, EventKmerPair* band_lower_left) {

    //fprintf(stderr, "%s\n", sequence);
    //fprintf(stderr, "Scaling %f %f", scaling.scale, scaling.shift);

    //size_t strand_idx = 0;
    //size_t k = 6;

    // size_t n_events = events[strand_idx].n;
    int32_t n_events = n_event;
    int32_t n_kmers = sequence_len - KMER_SIZE + 1;
    //fprintf(stderr,"n_kmers : %d\n",n_kmers);
    // backtrack markers
    //const uint8_t FROM_D = 0;
    //const uint8_t FROM_U = 1;
    //const uint8_t FROM_L = 2;

    // qc
    //double min_average_log_emission = -5.0;
    //int max_gap_threshold = 50;

    // banding
    //int bandwidth = ALN_BANDWIDTH;
    //half_bandwidth = bandwidth / 2;

    // transition penalties
    float events_per_kmer = (float)n_events / n_kmers;
    float p_stay = 1 - (1 / (events_per_kmer + 1));

    // setting a tiny skip penalty helps keep the true alignment within the adaptive band
    // this was empirically determined
    //double epsilon = 1e-10;
    double lp_skip = log(epsilon);
    double lp_stay = log(p_stay);
    double lp_step = log(1.0 - exp(lp_skip) - exp(lp_stay));
    double lp_trim = log(0.01);

    // dp matrix
    int32_t n_rows = n_events + 1;
    int32_t n_cols = n_kmers + 1;
    int32_t n_bands = n_rows + n_cols;
    //
    // Backtrack to compute alignment
    //
    double sum_emission = 0;
    double n_aligned_events = 0;

    //>>>>>>>>>>>>>> New replacement begin
    // std::vector<AlignedPair> out;

    int outIndex = 0;
    //<<<<<<<<<<<<<<<<New Replacement over

    float max_score = -INFINITY;
    int curr_event_idx = 0;
    int curr_kmer_idx = n_kmers - 1;

    // Find best score between an event and the last k-mer. after trimming the remaining evnets
    for (int32_t event_idx = 0; event_idx < n_events; ++event_idx) {
        int band_idx = event_kmer_to_band(event_idx, curr_kmer_idx);

        //>>>>>>>New  replacement begin
        /*assert(band_idx < bands.size());*/

        assert(band_idx < n_bands);

        //<<<<<<<<New Replacement over
        int offset = band_event_to_offset(band_idx, event_idx);
        if (is_offset_valid(offset)) {
            float s =
                BAND_ARRAY(band_idx,offset) + (n_events - event_idx) * lp_trim;
            if (s > max_score) {
                max_score = s;
                curr_event_idx = event_idx;
            }
        }
    }

#ifdef DEBUG_ADAPTIVE
    fprintf(stderr, "[adaback] ei: %d ki: %d s: %.2f\n", curr_event_idx,
            curr_kmer_idx, max_score);
#endif

    int curr_gap = 0;
    int max_gap = 0;
    while (curr_kmer_idx >= 0 && curr_event_idx >= 0) {
        // emit alignment
        //>>>>>>>New Repalcement begin
        assert(outIndex < n_events * 2);
        out_2[outIndex].ref_pos = curr_kmer_idx;
        out_2[outIndex].read_pos = curr_event_idx;
        outIndex++;
        // out.push_back({curr_kmer_idx, curr_event_idx});
        //<<<<<<<<<New Replacement over

#ifdef DEBUG_ADAPTIVE
        fprintf(stderr, "[adaback] ei: %d ki: %d\n", curr_event_idx,
                curr_kmer_idx);
#endif
        // qc stats
        //>>>>>>>>>>>>>>New Replacement begin
        char* substring = &sequence[curr_kmer_idx];
        int32_t kmer_rank = get_kmer_rank(substring, KMER_SIZE);
        //<<<<<<<<<<<<<New Replacement over
        float tempLogProb = log_probability_match_r9(
            scaling, models, events, curr_event_idx, kmer_rank);

        sum_emission += tempLogProb;
        //fprintf(stderr, "lp_emission %f \n", tempLogProb);
        //fprintf(stderr,"lp_emission %f, sum_emission %f, n_aligned_events %d\n",tempLogProb,sum_emission,outIndex);

        n_aligned_events += 1;

        int band_idx = event_kmer_to_band(curr_event_idx, curr_kmer_idx);
        int offset = band_event_to_offset(band_idx, curr_event_idx);
        assert(band_kmer_to_offset(band_idx, curr_kmer_idx) == offset);

        uint8_t from = TRACE_ARRAY(band_idx,offset);
        if (from == FROM_D) {
            curr_kmer_idx -= 1;
            curr_event_idx -= 1;
            curr_gap = 0;
        } else if (from == FROM_U) {
            curr_event_idx -= 1;
            curr_gap = 0;
        } else {
            curr_kmer_idx -= 1;
            curr_gap += 1;
            max_gap = MAX(curr_gap, max_gap);
        }
    }

    //>>>>>>>>New replacement begin
    // std::reverse(out.begin(), out.end());
    int c;
    int end = outIndex - 1;
    for (c = 0; c < outIndex / 2; c++) {
        int ref_pos_temp = out_2[c].ref_pos;
        int read_pos_temp = out_2[c].read_pos;
        out_2[c].ref_pos = out_2[end].ref_pos;
        out_2[c].read_pos = out_2[end].read_pos;
        out_2[end].ref_pos = ref_pos_temp;
        out_2[end].read_pos = read_pos_temp;
        end--;
    }

    // if(outIndex>1){
    //   AlignedPair temp={out_2[0].ref_pos,out[0].read_pos};
    //   int i;
    //   for(i=0;i<outIndex-1;i++){
    //     out_2[i]={out_2[outIndex-1-i].ref_pos,out[outIndex-1-i].read_pos};
    //   }
    //   out[outIndex-1]={temp.ref_pos,temp.read_pos};
    // }
    //<<<<<<<<<New replacement over

    // QC results
    double avg_log_emission = sum_emission / n_aligned_events;
    //fprintf(stderr,"sum_emission %f, n_aligned_events %f, avg_log_emission %f\n",sum_emission,n_aligned_events,avg_log_emission);
    //>>>>>>>>>>>>>New replacement begin
    bool spanned = out_2[0].ref_pos == 0 &&
                   out_2[outIndex - 1].ref_pos == int(n_kmers - 1);
    // bool spanned = out.front().ref_pos == 0 && out.back().ref_pos == n_kmers - 1;
    //<<<<<<<<<<<<<New replacement over
    //bool failed = false;
    if (avg_log_emission < min_average_log_emission || !spanned ||
        max_gap > max_gap_threshold) {
        //failed = true;
        //>>>>>>>>>>>>>New replacement begin
        outIndex = 0;
        // out.clear();
        //free(out_2);
        //out_2 = NULL;
        //<<<<<<<<<<<<<New replacement over
    }

    // free(kmer_ranks);
    // for (size_t i = 0; i < n_bands; i++) {
    //     free(bands[i]);
    //     free(trace[i]);
    // }
    // free(bands);
    // free(trace);
    // free(band_lower_left);
    //fprintf(stderr, "ada\t%s\t%s\t%.2lf\t%zu\t%.2lf\t%d\t%d\t%d\n", read.read_name.substr(0, 6).c_str(), failed ? "FAILED" : "OK", events_per_kmer, sequence.size(), avg_log_emission, curr_event_idx, max_gap, fills);
    //outSize=outIndex;
    //if(outIndex>500000)fprintf(stderr, "Max outSize %d\n", outIndex);
    return outIndex;
}


__global__ void align_kernel_pre(AlignedPair* event_align_pairs,
    int32_t* n_event_align_pairs, char* read,
    int32_t* read_len, int32_t* read_ptr,
    event_t* event_table, int32_t* n_events,
    int32_t* event_ptr, model_t* model,
    scalings_t* scalings, int32_t n_bam_rec,int32_t* kmer_ranks,float *bands,uint8_t *trace, EventKmerPair* band_lower_left) {

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < n_bam_rec) {
        AlignedPair* out_2 = &event_align_pairs[event_ptr[i] * 2];
        char* sequence = &read[read_ptr[i]];
        int32_t sequence_len = read_len[i];
        event_t* events = &event_table[event_ptr[i]];
        int32_t n_event = n_events[i];
        scalings_t scaling = scalings[i];
        int32_t* kmer_rank = &kmer_ranks[read_ptr[i]];
        float *band = &bands[(read_ptr[i]+event_ptr[i])*ALN_BANDWIDTH];
        uint8_t *trace1 = &trace[(read_ptr[i]+event_ptr[i])*ALN_BANDWIDTH];
        EventKmerPair* band_lower_left1 = &band_lower_left[read_ptr[i]+event_ptr[i]];;

        align_single_pre(out_2, sequence, sequence_len,
                            events, n_event, model, scaling,kmer_rank,band,trace1,band_lower_left1);
    }
}


__global__ void 
//__launch_bounds__(MY_KERNEL_MAX_THREADS, MY_KERNEL_MIN_BLOCKS)
align_kernel_core(AlignedPair* event_align_pairs,
    int32_t* n_event_align_pairs, char* read,
    int32_t* read_len, int32_t* read_ptr,
    event_t* event_table, int32_t* n_events,
    int32_t* event_ptr, model_t* model,
    scalings_t* scalings, int32_t n_bam_rec,int32_t* kmer_ranks,float *bands,uint8_t *trace, EventKmerPair* band_lower_left) {

#ifndef WARP_HACK        
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n_bam_rec) {
#else
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int i = tid/32;    
    if (i < n_bam_rec && tid%32==0) {
#endif        
        AlignedPair* out_2 = &event_align_pairs[event_ptr[i] * 2];
        char* sequence = &read[read_ptr[i]];
        int32_t sequence_len = read_len[i];
        event_t* events = &event_table[event_ptr[i]];
        int32_t n_event = n_events[i];
        scalings_t scaling = scalings[i];
        int32_t* kmer_rank = &kmer_ranks[read_ptr[i]];
        float *band = &bands[(read_ptr[i]+event_ptr[i])*ALN_BANDWIDTH];
        uint8_t *trace1 = &trace[(read_ptr[i]+event_ptr[i])*ALN_BANDWIDTH];
        EventKmerPair* band_lower_left1 = &band_lower_left[read_ptr[i]+event_ptr[i]];;

        align_single(out_2, sequence, sequence_len,
                            events, n_event, model, scaling,kmer_rank,band,trace1,band_lower_left1);
    }
}


__global__ void align_kernel_post(AlignedPair* event_align_pairs,
    int32_t* n_event_align_pairs, char* read,
    int32_t* read_len, int32_t* read_ptr,
    event_t* event_table, int32_t* n_events,
    int32_t* event_ptr, model_t* model,
    scalings_t* scalings, int32_t n_bam_rec,int32_t* kmer_ranks,float *bands,uint8_t *trace, EventKmerPair* band_lower_left) {

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < n_bam_rec) {
        AlignedPair* out_2 = &event_align_pairs[event_ptr[i] * 2];
        char* sequence = &read[read_ptr[i]];
        int32_t sequence_len = read_len[i];
        event_t* events = &event_table[event_ptr[i]];
        int32_t n_event = n_events[i];
        scalings_t scaling = scalings[i];
        int32_t* kmer_rank = &kmer_ranks[read_ptr[i]];
        float *band = &bands[(read_ptr[i]+event_ptr[i])*ALN_BANDWIDTH];
        uint8_t *trace1 = &trace[(read_ptr[i]+event_ptr[i])*ALN_BANDWIDTH];
        EventKmerPair* band_lower_left1 = &band_lower_left[read_ptr[i]+event_ptr[i]];;

        n_event_align_pairs[i] = align_single_post(out_2, sequence, sequence_len,
                            events, n_event, model, scaling,kmer_rank,band,trace1,band_lower_left1);
    }
}



__global__ void 
//__launch_bounds__(MY_KERNEL_MAX_THREADS, MY_KERNEL_MIN_BLOCKS)
align_kernel_core_2d(AlignedPair* event_align_pairs,
    int32_t* n_event_align_pairs, char* read,
    int32_t* read_len, int32_t* read_ptr,
    event_t* event_table, int32_t* n_events1,
    int32_t* event_ptr, model_t* models,
    scalings_t* scalings, int32_t n_bam_rec,int32_t* kmer_rank,float *band,uint8_t *traces, EventKmerPair* band_lower_lefts) {
   
    int i = blockDim.y * blockIdx.y + threadIdx.y;
    int tid=blockIdx.x*blockDim.x+threadIdx.x;

    if (i < n_bam_rec) {   

        AlignedPair* out_2 = &event_align_pairs[event_ptr[i] * 2];
        char* sequence = &read[read_ptr[i]];
        int32_t sequence_len = read_len[i];
        event_t* events = &event_table[event_ptr[i]];
        int32_t n_event = n_events1[i];
        scalings_t scaling = scalings[i];
        int32_t* kmer_ranks = &kmer_rank[read_ptr[i]];
        float *bands = &band[(read_ptr[i]+event_ptr[i])*ALN_BANDWIDTH];
        uint8_t *trace = &traces[(read_ptr[i]+event_ptr[i])*ALN_BANDWIDTH];
        EventKmerPair* band_lower_left = &band_lower_lefts[read_ptr[i]+event_ptr[i]];;

        // size_t n_events = events[strand_idx].n;
        int32_t n_events = n_event;
        int32_t n_kmers = sequence_len - KMER_SIZE + 1;
        //fprintf(stderr,"n_kmers : %d\n",n_kmers);

        // transition penalties
        float events_per_kmer = (float)n_events / n_kmers;
        float p_stay = 1 - (1 / (events_per_kmer + 1));

        // setting a tiny skip penalty helps keep the true alignment within the adaptive band
        // this was empirically determined
        //double epsilon = 1e-10;
        double lp_skip = log(epsilon);
        double lp_stay = log(p_stay);
        double lp_step = log(1.0 - exp(lp_skip) - exp(lp_stay));
        double lp_trim = log(0.01);

        // dp matrix
        int32_t n_rows = n_events + 1;
        int32_t n_cols = n_kmers + 1;
        int32_t n_bands = n_rows + n_cols;                                    

        // fill in remaining bands
        for (int32_t band_idx = 2; band_idx < n_bands; ++band_idx) {
            // Determine placement of this band according to Suzuki's adaptive algorithm
            // When both ll and ur are out-of-band (ob) we alternate movements
            // otherwise we decide based on scores
            float ll = BAND_ARRAY((band_idx - 1), 0);
            float ur = BAND_ARRAY((band_idx - 1),(bandwidth - 1));
            bool ll_ob = ll == -INFINITY;
            bool ur_ob = ur == -INFINITY;

            bool right = false;
            if (ll_ob && ur_ob) {
                right = band_idx % 2 == 1;
            } else {
                right = ll < ur; // Suzuki's rule
            }

            if (right) {
                band_lower_left[band_idx] =
                    move_right(band_lower_left[band_idx - 1]);
            } else {
                band_lower_left[band_idx] =
                    move_down(band_lower_left[band_idx - 1]);
            }
            // If the trim state is within the band, fill it in here
            int trim_offset = band_kmer_to_offset(band_idx, -1);
            if (is_offset_valid(trim_offset)) {
                int32_t event_idx = event_at_offset(band_idx, trim_offset);
                if (event_idx >= 0 && event_idx < n_events) {
                    BAND_ARRAY(band_idx,trim_offset) = lp_trim * (event_idx + 1);
                    TRACE_ARRAY(band_idx,trim_offset) = FROM_U;
                } else {
                    BAND_ARRAY(band_idx,trim_offset) = -INFINITY;
                }
            }

            // Get the offsets for the first and last event and kmer
            // We restrict the inner loop to only these values
            int kmer_min_offset = band_kmer_to_offset(band_idx, 0);
            int kmer_max_offset = band_kmer_to_offset(band_idx, n_kmers);
            int event_min_offset = band_event_to_offset(band_idx, n_events - 1);
            int event_max_offset = band_event_to_offset(band_idx, -1);

            int min_offset = MAX(kmer_min_offset, event_min_offset);
            min_offset = MAX(min_offset, 0);

            int max_offset = MIN(kmer_max_offset, event_max_offset);
            max_offset = MIN(max_offset, bandwidth);

            __syncthreads();    
   
            if(tid < (max_offset-min_offset)) {
                int offset=min_offset+tid;

                int event_idx = event_at_offset(band_idx, offset);
                int kmer_idx = kmer_at_offset(band_idx, offset);

                int32_t kmer_rank = kmer_ranks[kmer_idx];

                int offset_up = band_event_to_offset(band_idx - 1, event_idx - 1);
                int offset_left = band_kmer_to_offset(band_idx - 1, kmer_idx - 1);
                int offset_diag = band_kmer_to_offset(band_idx - 2, kmer_idx - 1);

    #ifdef DEBUG_ADAPTIVE
                // verify loop conditions
                assert(kmer_idx >= 0 && kmer_idx < n_kmers);
                assert(event_idx >= 0 && event_idx < n_events);
                assert(offset_diag ==
                       band_event_to_offset(band_idx - 2, event_idx - 1));
                assert(offset_up - offset_left == 1);
                assert(offset >= 0 && offset < bandwidth);
    #endif //DEBUG_ADAPTIVE

                float up = is_offset_valid(offset_up)
                               ? BAND_ARRAY(band_idx - 1,offset_up)
                               : -INFINITY;
                float left = is_offset_valid(offset_left)
                                 ? BAND_ARRAY(band_idx - 1,offset_left)
                                 : -INFINITY;
                float diag = is_offset_valid(offset_diag)
                                 ? BAND_ARRAY(band_idx - 2,offset_diag)
                                 : -INFINITY;

                float lp_emission = log_probability_match_r9(
                    scaling, models, events, event_idx, kmer_rank);
                //fprintf(stderr, "lp emiision : %f , event idx %d, kmer rank %d\n", lp_emission,event_idx,kmer_rank);
                float score_d = diag + lp_step + lp_emission;
                float score_u = up + lp_stay + lp_emission;
                float score_l = left + lp_skip;

                float max_score = score_d;
                uint8_t from = FROM_D;

                max_score = score_u > max_score ? score_u : max_score;
                from = max_score == score_u ? FROM_U : from;
                max_score = score_l > max_score ? score_l : max_score;
                from = max_score == score_l ? FROM_L : from;

    #ifdef DEBUG_ADAPTIVE
                fprintf(stderr,
                        "[adafill] offset-up: %d offset-diag: %d offset-left: %d\n",
                        offset_up, offset_diag, offset_left);
                fprintf(stderr, "[adafill] up: %.2lf diag: %.2lf left: %.2lf\n", up,
                        diag, left);
                fprintf(stderr,
                        "[adafill] bi: %d o: %d e: %d k: %d s: %.2lf f: %d emit: "
                        "%.2lf\n",
                        band_idx, offset, event_idx, kmer_idx, max_score, from,
                        lp_emission);
    #endif //DEBUG_ADAPTIVE
                BAND_ARRAY(band_idx,offset) = max_score;
                TRACE_ARRAY(band_idx,offset) = from;
                //fills += 1;
            }

            __syncthreads();  
   
        }

    }

}



#endif