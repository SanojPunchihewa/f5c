#include <errno.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>

#include "error.h"
#include "f5c.h"
#include "f5cmisc.cuh"
#include "f5cmisc.h"


//#define BLOCK_LEN 8


#ifdef CONST_MEM
  __constant__ model_t model[4096];
#endif


void align_cuda(core_t* core, db_t* db) {
    int32_t i;
    int32_t n_bam_rec = db->n_bam_rec;

    /**allocate and copy input arrays for cuda*/

    char* read;        //flattened reads sequences
    int32_t* read_ptr; //index pointer for flattedned "reads"
    int32_t* read_len;
    int64_t sum_read_len;
    //get the total size and create the pointers
    int32_t* read_ptr_host = (int32_t*)malloc(sizeof(int32_t) * n_bam_rec);
    MALLOC_CHK(read_ptr_host);
    sum_read_len = 0;

    //read sequences : needflattening
    for (i = 0; i < n_bam_rec; i++) {
        read_ptr_host[i] = sum_read_len;
        sum_read_len += (db->read_len[i] + 1); //with null term
    }
    //form the temporary flattened array on host
    char* read_host = (char*)malloc(sizeof(char) * sum_read_len);
    MALLOC_CHK(read_host);
    for (i = 0; i < n_bam_rec; i++) {
        int32_t idx = read_ptr_host[i];
        strcpy(&read_host[idx], db->read[i]);
    }

    //copy to the gpu
    print_size("read_ptr array",n_bam_rec * sizeof(int32_t));
    hipMalloc((void**)&read_ptr, n_bam_rec * sizeof(int32_t));
    CUDA_CHK();
    hipMemcpy(read_ptr, read_ptr_host, n_bam_rec * sizeof(int32_t),
               hipMemcpyHostToDevice);
    CUDA_CHK();
    print_size("read array",sum_read_len * sizeof(char));
    hipMalloc((void**)&read, sum_read_len * sizeof(char)); //with null char
    CUDA_CHK();
    hipMemcpy(read, read_host, sum_read_len * sizeof(char),
               hipMemcpyHostToDevice);
    CUDA_CHK();

    //read length : already linear hence direct copy
    print_size("read_lens",n_bam_rec * sizeof(int32_t));
    hipMalloc((void**)&read_len, n_bam_rec * sizeof(int32_t));
    CUDA_CHK();
    hipMemcpy(read_len, db->read_len, n_bam_rec * sizeof(int32_t),
               hipMemcpyHostToDevice);
    CUDA_CHK();

    //now the events : need flattening

    int32_t* n_events;
    event_t* event_table;
    int32_t* event_ptr;
    int64_t sum_n_events;

    //num events : need flattening
    //get the total size and create the pointers
    int32_t* n_events_host = (int32_t*)malloc(sizeof(int32_t) * n_bam_rec);
    MALLOC_CHK(n_events_host);
    int32_t* event_ptr_host = (int32_t*)malloc(sizeof(int32_t) * n_bam_rec);
    MALLOC_CHK(event_ptr_host);

    sum_n_events = 0;
    for (i = 0; i < n_bam_rec; i++) {
        n_events_host[i] = db->et[i].n;
        event_ptr_host[i] = sum_n_events;
        sum_n_events += db->et[i].n;
    }

    //n_events copy
    print_size("n_events",n_bam_rec * sizeof(int32_t));
    hipMalloc((void**)&n_events, n_bam_rec * sizeof(int32_t));
    CUDA_CHK();
    hipMemcpy(n_events, n_events_host, n_bam_rec * sizeof(int32_t),
               hipMemcpyHostToDevice);
    CUDA_CHK();
    //event ptr copy
    print_size("event ptr",n_bam_rec * sizeof(int32_t));
    hipMalloc((void**)&event_ptr, n_bam_rec * sizeof(int32_t));
    CUDA_CHK();
    hipMemcpy(event_ptr, event_ptr_host, n_bam_rec * sizeof(int32_t),
               hipMemcpyHostToDevice);
    CUDA_CHK();

    //event table flatten
    //form the temporary flattened array on host
    event_t* event_table_host =
        (event_t*)malloc(sizeof(event_t) * sum_n_events);
    MALLOC_CHK(event_table_host);
    for (i = 0; i < n_bam_rec; i++) {
        int32_t idx = event_ptr_host[i];
        memcpy(&event_table_host[idx], db->et[i].event,
               sizeof(event_t) * db->et[i].n);
    }

    print_size("event table",sum_n_events * sizeof(event_t));
    hipMalloc((void**)&event_table, sum_n_events * sizeof(event_t));
    CUDA_CHK();
    hipMemcpy(event_table, event_table_host, sizeof(event_t) * sum_n_events,
               hipMemcpyHostToDevice);
    CUDA_CHK();

    //model : already linear
#ifndef CONST_MEM
    model_t* model;
    hipMalloc((void**)&model,
               NUM_KMER * sizeof(model_t)); //todo : constant memory
    CUDA_CHK();
    hipMemcpy(model, core->model, NUM_KMER * sizeof(model_t),
               hipMemcpyHostToDevice);
    CUDA_CHK();
#else
    hipMemcpyToSymbol(HIP_SYMBOL(model),  core->model, NUM_KMER * sizeof(model_t));
#endif

    //scalings : already linear
    scalings_t* scalings;
    print_size("Scalings",n_bam_rec * sizeof(scalings_t));
    hipMalloc((void**)&scalings, n_bam_rec * sizeof(scalings_t));
    CUDA_CHK();
    hipMemcpy(scalings, db->scalings, sizeof(scalings_t) * n_bam_rec,
               hipMemcpyHostToDevice);
    CUDA_CHK();

    /**allocate output arrays for cuda**/
    AlignedPair* event_align_pairs;
    int32_t* n_event_align_pairs;
    print_size("event align pairs",2 * sum_n_events *sizeof(AlignedPair));
    hipMalloc((void**)&event_align_pairs,
               2 * sum_n_events *
                   sizeof(AlignedPair)); //todo : need better huristic
    CUDA_CHK();
    print_size("n_event_align_pairs",n_bam_rec * sizeof(int32_t));
    hipMalloc((void**)&n_event_align_pairs, n_bam_rec * sizeof(int32_t));
    CUDA_CHK();

    //scratch arrays

    size_t sum_n_bands = sum_n_events + sum_read_len; //todo : can be optimised

    int32_t *kmer_ranks;
    float *bands;
    uint8_t *trace;
    EventKmerPair* band_lower_left;

    print_size("kmer ranks",sizeof(int32_t) * sum_read_len);
    hipMalloc((void**)&kmer_ranks,sizeof(int32_t) * sum_read_len); //todo : optimise by the sum of n_kmers
    CUDA_CHK();
    print_size("bands",sizeof(float) * sum_n_bands * ALN_BANDWIDTH);
    hipMalloc((void**)&bands,sizeof(float) * sum_n_bands * ALN_BANDWIDTH);
    CUDA_CHK();
    print_size("trace",sizeof(uint8_t) * sum_n_bands * ALN_BANDWIDTH);
    hipMalloc((void**)&trace, sizeof(uint8_t) * sum_n_bands * ALN_BANDWIDTH);
    CUDA_CHK();
    print_size("band_lower_left",sizeof(EventKmerPair)* sum_n_bands);
    hipMalloc((void**)&band_lower_left, sizeof(EventKmerPair)* sum_n_bands);
    CUDA_CHK();

    //cuda kernel configuraion parameters
    int32_t BLOCK_LEN = core->opt.cuda_block_size;
    dim3 grid((db->n_bam_rec + BLOCK_LEN - 1) / BLOCK_LEN);
    dim3 block(BLOCK_LEN);

    fprintf(stderr,"grid %d, block %d\n",grid.x,block.x);
    // hipDeviceSetLimit(hipLimitMallocHeapSize, 512 * 1024 * 1024);
    // CUDA_CHK();

#ifndef CONST_MEM

    #ifndef ALIGN_KERNEL_SLICED
        align_kernel<<<grid, block>>>(event_align_pairs, n_event_align_pairs, read,
                                  read_len, read_ptr, event_table, n_events,
                                  event_ptr, model, scalings, n_bam_rec, kmer_ranks,bands,trace,band_lower_left );
    #else
        double realtime1 = realtime();    
        #ifndef TWODIM_ALIGN_PRE  
            align_kernel_pre<<<grid, block>>>(event_align_pairs, n_event_align_pairs, read,
            read_len, read_ptr, event_table, n_events,
            event_ptr, model, scalings, n_bam_rec, kmer_ranks,bands,trace,band_lower_left );
        #else
            // #ifdef  PRE_3D    
            //     dim3 gridpre((sum_n_bands + BLOCK_LEN_NUMBAND - 1) / BLOCK_LEN_NUMBAND,(),(db->n_bam_rec + BLOCK_LEN_READS2 - 1) / BLOCK_LEN_READS2);
            //     dim3 blockpre(BLOCK_LEN_NUMBAND,BLOCK_LEN_BANDWIDTH3,BLOCK_LEN_READS2);    
            // #else
            //     dim3 gridpre((sum_n_bands + BLOCK_LEN_NUMBAND - 1) / BLOCK_LEN_NUMBAND,(db->n_bam_rec + BLOCK_LEN_READS2 - 1) / BLOCK_LEN_READS2);
            //     dim3 blockpre(BLOCK_LEN_NUMBAND,BLOCK_LEN_READS2);    
            // #endif
            assert(BLOCK_LEN_BANDWIDTH>=ALN_BANDWIDTH);
            dim3 gridpre(1,(db->n_bam_rec + BLOCK_LEN_READS - 1) / BLOCK_LEN_READS);
            dim3 blockpre(BLOCK_LEN_BANDWIDTH,BLOCK_LEN_READS);           
            align_kernel_pre_2d<<<gridpre, blockpre>>>(event_align_pairs, n_event_align_pairs, read,
            read_len, read_ptr, event_table, n_events,
            event_ptr, model, scalings, n_bam_rec, kmer_ranks,bands,trace,band_lower_left );
        #endif
        hipDeviceSynchronize();CUDA_CHK();
        fprintf(stderr, "[%s::%.3f*%.2f] align pre done\n", __func__,
                realtime() - realtime1, cputime() / (realtime() - realtime1));
                
        realtime1 = realtime();

        #ifndef TWODIM_ALIGN_CORE    
            #ifndef WARP_HACK      
                align_kernel_core<<<grid, block>>>(event_align_pairs, n_event_align_pairs, read,
                    read_len, read_ptr, event_table, n_events,
                    event_ptr, model, scalings, n_bam_rec, kmer_ranks,bands,trace,band_lower_left );
            #else
                assert(BLOCK_LEN>=32);    
                dim3 grid1((db->n_bam_rec + (BLOCK_LEN/32) - 1) / (BLOCK_LEN/32)); 
                fprintf(stderr,"grid new %d\n",grid1.x);   
                align_kernel_core<<<grid1, block>>>(event_align_pairs, n_event_align_pairs, read,
                read_len, read_ptr, event_table, n_events,
                event_ptr, model, scalings, n_bam_rec, kmer_ranks,bands,trace,band_lower_left );   
            #endif
        #else
            assert(BLOCK_LEN_BANDWIDTH>=ALN_BANDWIDTH);
            dim3 grid1(1,(db->n_bam_rec + BLOCK_LEN_READS - 1) / BLOCK_LEN_READS);
            dim3 block1(BLOCK_LEN_BANDWIDTH,BLOCK_LEN_READS);
            align_kernel_core_2d<<<grid1, block1>>>(event_align_pairs, n_event_align_pairs, read,
                    read_len, read_ptr, event_table, n_events,
                    event_ptr, model, scalings, n_bam_rec, kmer_ranks,bands,trace,band_lower_left );

        #endif        



        hipDeviceSynchronize();CUDA_CHK();
        fprintf(stderr, "[%s::%.3f*%.2f] align done\n", __func__,
        realtime() - realtime1, cputime() / (realtime() - realtime1));
            
        realtime1 = realtime();
        align_kernel_post<<<grid, block>>>(event_align_pairs, n_event_align_pairs, read,
                read_len, read_ptr, event_table, n_events,
                event_ptr, model, scalings, n_bam_rec, kmer_ranks,bands,trace,band_lower_left );
        hipDeviceSynchronize();CUDA_CHK();
        fprintf(stderr, "[%s::%.3f*%.2f] align post done\n", __func__,
                realtime() - realtime1, cputime() / (realtime() - realtime1));

    #endif  

#else
    align_kernel<<<grid, block>>>(event_align_pairs, n_event_align_pairs, read,
                                read_len, read_ptr, event_table, n_events,
                                event_ptr, scalings, n_bam_rec, kmer_ranks,bands,trace,band_lower_left );
#endif

    //fprintf(stderr,"readlen %d,n_events %d\n",db->read_len[i],n_event_align_pairs);

#ifdef CUDA_DEBUG

    hipDeviceSynchronize();
    hipError_t code = hipGetLastError();
    //todo : print a message to detect the launch timed out
    if (code == hipErrorLaunchTimeOut) {
        ERROR("%s", "The kernel timed out. You have to first disable the cuda "
                    "time out.");
        fprintf(
            stderr,
            "On Ubuntu do the following\nOpen the file /etc/X11/xorg.conf\nYou "
            "will have a section about your NVIDIA device. Add the following "
            "line to it.\nOption \"Interactive\" \"0\"\nIf you do not have a "
            "section about your NVIDIA device in /etc/X11/xorg.conf or you do "
            "not have a file named /etc/X11/xorg.conf, run the command sudo "
            "nvidia-xconfig to generate a xorg.conf file and do as above.\n\n");
    }
    if (code != hipSuccess) {
        fprintf(stderr, "Cuda error: %s \n in file : %s line number : %lu\n",
                hipGetErrorString(code), __FILE__, __LINE__);
        exit(-1);
    }        
    
#endif

    //copyback ans
    hipMemcpy(db->n_event_align_pairs, n_event_align_pairs,
               n_bam_rec * sizeof(int32_t), hipMemcpyDeviceToHost);
    CUDA_CHK();
    AlignedPair* event_align_pairs_host =
        (AlignedPair*)malloc(2 * sum_n_events * sizeof(AlignedPair));
    MALLOC_CHK(event_align_pairs_host);
    hipMemcpy(event_align_pairs_host, event_align_pairs,
               2 * sum_n_events * sizeof(AlignedPair), hipMemcpyDeviceToHost);
    CUDA_CHK();
    //copy back
    for (i = 0; i < n_bam_rec; i++) {
        int32_t idx = event_ptr_host[i];
        memcpy(db->event_align_pairs[i], &event_align_pairs_host[idx * 2],
               sizeof(AlignedPair) * db->n_event_align_pairs[i]);
    }

    //free the temp arrays on host
    free(read_host);
    free(read_ptr_host);
    free(n_events_host);
    free(event_ptr_host);
    free(event_table_host);
    free(event_align_pairs_host);
    hipFree(read_ptr);
    hipFree(read); //with null char
    hipFree(read_len);
    hipFree(n_events);
    hipFree(event_ptr);
    hipFree(event_table);
    hipFree(model); //constant memory
    hipFree(scalings);
    hipFree(event_align_pairs);
    hipFree(n_event_align_pairs);
    hipFree(kmer_ranks);
    hipFree(bands);
    hipFree(trace);
    hipFree(band_lower_left);


}
